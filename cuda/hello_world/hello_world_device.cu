// UCSD Phys244
// Spring 2018
// Andreas Goetz (agoetz@sdsc.edu)

// Hello World Program in CUDA C
//
// Contains a function that is executed on the device (GPU)
//


#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void my_kernel(void){
}

int main(void) {

  my_kernel<<<1,1>>>();
  printf("Hello World!\n");
  return 0;

}
