/*
* Copyright 1993-2010 NVIDIA Corporation. All rights reserved. *
* NVIDIA Corporation and its licensors retain all intellectual property and
*
* Updated by Mary Thomas, April 2023, for simple cuda compile example 
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void kernel( void ) { }
int main( void ) { kernel<<<1,1>>>();
printf( "Hello,  SDSC HPC Training World!\n" ); return 0;
}
